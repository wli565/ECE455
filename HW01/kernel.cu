#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vector_add(const float* a, const float* b, float* c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int n = 16;
    size_t size = n * sizeof(float);

    float h_a[16], h_b[16], h_c[16];
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void **)&d_c, size);
    
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    vector_add<<<1, n>>>(d_a, d_b, d_c, n);

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    std::cout << "Result: ";
    for (int i = 0; i < n; i++) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
6
