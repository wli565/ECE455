
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstddef>
#include <cstdint>
#include <iomanip>
#include <iostream>
#include <random>
#include <stdexcept>
#include <vector>

#define BLOCK_DIM 32
#define MAT_DIM 1024 

#define checkCuda(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template <typename T>
std::vector<T> create_rand_vector(size_t n)
{
    std::random_device r;
    std::default_random_engine e(r());
    std::uniform_int_distribution<int> uniform_dist(-256, 256);

    std::vector<T> vec(n);
    for (size_t i{0}; i < n; ++i)
    {
        vec.at(i) = static_cast<T>(uniform_dist(e));
    }

    return vec;
}

// mat_1: m x n
// mat_2: n x p
// mat_3: m x p
template <typename T>
void mm(T const* mat_1, T const* mat_2, T* mat_3, size_t m, size_t n, size_t p)
{
    for (size_t i{0}; i < m; ++i)
    {
        for (size_t j{0}; j < p; ++j)
        {
            T acc_sum{0};
            for (size_t k{0}; k < n; ++k)
            {
                acc_sum += mat_1[i * n + k] * mat_2[k * p + j];
            }
            mat_3[i * p + j] = acc_sum;
        }
    }
}

// ------------------- Loop-Unrolled Kernel -------------------
template <typename T>
__global__ void mm_unrolled_kernel(T const* mat_1, T const* mat_2, T* mat_3,
                                   size_t m, size_t n, size_t p)
{
    size_t j{blockIdx.x * blockDim.x + threadIdx.x};
    size_t i{blockIdx.y * blockDim.y + threadIdx.y};

    if ((i >= m) || (j >= p))
    {
        return;
    }

    T acc_sum{0};
    size_t k{0};
    for (; k + 3 < n; k += 4)
    {
        acc_sum += mat_1[i * n + (k + 0)] * mat_2[(k + 0) * p + j];
        acc_sum += mat_1[i * n + (k + 1)] * mat_2[(k + 1) * p + j];
        acc_sum += mat_1[i * n + (k + 2)] * mat_2[(k + 2) * p + j];
        acc_sum += mat_1[i * n + (k + 3)] * mat_2[(k + 3) * p + j];
    }
    for (; k < n; ++k)
    {
        acc_sum += mat_1[i * n + k] * mat_2[k * p + j];
    }

    mat_3[i * p + j] = acc_sum;
}

// ------------------- Launcher -------------------
template <typename T>
void mm_cuda(T const* mat_1, T const* mat_2, T* mat_3,
             size_t m, size_t n, size_t p)
{
    dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
    dim3 blocks_per_grid(1, 1);
    blocks_per_grid.x = std::ceil(static_cast<double>(p) /
                                  static_cast<double>(threads_per_block.x));
    blocks_per_grid.y = std::ceil(static_cast<double>(m) /
                                  static_cast<double>(threads_per_block.y));
    mm_unrolled_kernel<<<blocks_per_grid, threads_per_block>>>(mat_1, mat_2, mat_3,
                                                               m, n, p);
}

// ------------------- Validation -------------------
template <typename T>
bool allclose(std::vector<T> const& vec_1, std::vector<T> const& vec_2,
              T const& abs_tol)
{
    if (vec_1.size() != vec_2.size())
    {
        return false;
    }
    for (size_t i{0}; i < vec_1.size(); ++i)
    {
        if (std::abs(vec_1.at(i) - vec_2.at(i)) > abs_tol)
        {
            std::cout << vec_1.at(i) << " " << vec_2.at(i) << std::endl;
            return false;
        }
    }
    return true;
}

template <typename T>
bool random_test_mm_cuda(size_t m, size_t n, size_t p)
{
    std::vector<T> const mat_1_vec{create_rand_vector<T>(m * n)};
    std::vector<T> const mat_2_vec{create_rand_vector<T>(n * p)};
    std::vector<T> mat_3_vec(m * p);
    std::vector<T> mat_4_vec(m * p);
    T const* mat_1{mat_1_vec.data()};
    T const* mat_2{mat_2_vec.data()};
    T* mat_3{mat_3_vec.data()};
    T* mat_4{mat_4_vec.data()};

    mm(mat_1, mat_2, mat_3, m, n, p);

    T *d_mat_1, *d_mat_2, *d_mat_4;
    checkCuda(hipMalloc(&d_mat_1, sizeof(T) * mat_1_vec.size()));
    checkCuda(hipMalloc(&d_mat_2, sizeof(T) * mat_2_vec.size()));
    checkCuda(hipMalloc(&d_mat_4, sizeof(T) * mat_4_vec.size()));

    checkCuda(hipMemcpy(d_mat_1, mat_1, sizeof(T) * mat_1_vec.size(),
                         hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_mat_2, mat_2, sizeof(T) * mat_2_vec.size(),
                         hipMemcpyHostToDevice));

    mm_cuda(d_mat_1, d_mat_2, d_mat_4, m, n, p);
    hipDeviceSynchronize();

    checkCuda(hipMemcpy(mat_4, d_mat_4, sizeof(T) * mat_4_vec.size(),
                         hipMemcpyDeviceToHost));

    checkCuda(hipFree(d_mat_1));
    checkCuda(hipFree(d_mat_2));
    checkCuda(hipFree(d_mat_4));

    return allclose<T>(mat_3_vec, mat_4_vec, 1e-4);
}

template <typename T>
bool random_multiple_test_mm_cuda(size_t num_tests)
{
    size_t m{MAT_DIM}, n{MAT_DIM}, p{MAT_DIM};
    bool success{false};
    for (size_t i{0}; i < num_tests; ++i)
    {
        success = random_test_mm_cuda<T>(m, n, p);
        if (!success)
        {
            return false;
        }
    }
    return true;
}

// ------------------- Benchmark -------------------
template <typename T>
float measure_latency_mm_cuda(size_t m, size_t n, size_t p,
                              size_t num_tests, size_t num_warmups)
{
    hipEvent_t startEvent, stopEvent;
    float time{0.0f};

    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));

    T *d_mat_1, *d_mat_2, *d_mat_4;
    checkCuda(hipMalloc(&d_mat_1, sizeof(T) * m * n));
    checkCuda(hipMalloc(&d_mat_2, sizeof(T) * n * p));
    checkCuda(hipMalloc(&d_mat_4, sizeof(T) * m * p));

    for (size_t i{0}; i < num_warmups; ++i)
    {
        mm_cuda(d_mat_1, d_mat_2, d_mat_4, m, n, p);
    }

    checkCuda(hipEventRecord(startEvent, 0));
    for (size_t i{0}; i < num_tests; ++i)
    {
        mm_cuda(d_mat_1, d_mat_2, d_mat_4, m, n, p);
    }
    checkCuda(hipEventRecord(stopEvent, 0));
    checkCuda(hipEventSynchronize(stopEvent));
    checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));

    checkCuda(hipFree(d_mat_1));
    checkCuda(hipFree(d_mat_2));
    checkCuda(hipFree(d_mat_4));

    return time / num_tests;
}

// ------------------- Main -------------------
int main()
{
    const size_t num_tests{2};
    assert(random_multiple_test_mm_cuda<int32_t>(num_tests));
    assert(random_multiple_test_mm_cuda<float>(num_tests));
    assert(random_multiple_test_mm_cuda<double>(num_tests));
    
    std::cout << "All tests passed!\n";

    const size_t num_measurement_tests{2};
    const size_t num_measurement_warmups{1};
    size_t m{MAT_DIM}, n{MAT_DIM}, p{MAT_DIM};

    float mm_cuda_int32_latency = measure_latency_mm_cuda<int32_t>(
        m, n, p, num_measurement_tests, num_measurement_warmups);
    float mm_cuda_float_latency = measure_latency_mm_cuda<float>(
        m, n, p, num_measurement_tests, num_measurement_warmups);
    float mm_cuda_double_latency = measure_latency_mm_cuda<double>(
        m, n, p, num_measurement_tests, num_measurement_warmups);

    std::cout << "Matrix Multiplication Runtime\n";
    std::cout << "m: " << m << " "
              << "n: " << n << " "
              << "p: " << p << "\n";
    std::cout << "INT32: " << std::fixed << std::setprecision(5)
              << mm_cuda_int32_latency << " ms\n";
    std::cout << "FLOAT: " << std::fixed << std::setprecision(5)
              << mm_cuda_float_latency << " ms\n";
    std::cout << "DOUBLE: " << std::fixed << std::setprecision(5)
              << mm_cuda_double_latency << " ms\n";

    return 0;
}