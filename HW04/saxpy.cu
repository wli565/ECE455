
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i] = a * x[i] + y[i];
}

int main() {
    int N = 1000000;
    size_t size = N * sizeof(float);
    float *x, *y, *d_x, *d_y;

    x = (float*)malloc(size);
    y = (float*)malloc(size);
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    saxpy<<<blocksPerGrid, threadsPerBlock>>>(N, 2.0f, d_x, d_y);

    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
    printf("y[0] = %f\n", y[0]);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);

    return 0;
}
