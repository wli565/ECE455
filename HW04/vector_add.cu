#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vector_add(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

int main() {
    int N = 100000;
    size_t size = N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("C[0] = %f\n", h_C[0]);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
