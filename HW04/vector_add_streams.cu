
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

int main() {
    int N = 1000000;
    size_t size = N * sizeof(float);
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);
    for (int i = 0; i < N; i++) { A[i] = 1.0f; B[i] = 2.0f; }

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    int half = N / 2;
    size_t half_size = size / 2;

    hipMemcpyAsync(d_A, A, half_size, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_B, B, half_size, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_A + half, A + half, half_size, hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(d_B + half, B + half, half_size, hipMemcpyHostToDevice, stream2);

    int threads = 256;
    int blocks_half = (half + threads - 1) / threads;
    vector_add<<<blocks_half, threads, 0, stream1>>>(d_A, d_B, d_C, half);
    vector_add<<<blocks_half, threads, 0, stream2>>>(d_A + half, d_B + half, d_C + half, half);

    hipMemcpyAsync(C, d_C, half_size, hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(C + half, d_C + half, half_size, hipMemcpyDeviceToHost, stream2);

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    printf("C[0] = %f, C[N-1] = %f\n", C[0], C[N-1]);

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(A); free(B); free(C);
    return 0;
}
